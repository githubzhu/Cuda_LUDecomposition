#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <stdlib.h>
#include <sys/time.h>
float *A,*L,*U,*input;
void arrayInit(int n);
void verifyLU(int n);
void updateLU(int n);
void freemem(int n);

__global__ void scale( float *a, int size, int c) {
	int index=c,k=0;//size=b

		for(k=index+1;k<size;k++) {
			a[size*index + k] = (float) a[size*index + k] / a[size*index + index];
		}

}
__global__ void reduce( float *a, int size, int c) {
	int tid = blockIdx.x;	//Handle the data at the index
	int index=c,j=0;//size=b

	for(j=index+1;j<size;j++) {
		a[((tid+index+1)*size + j)] = (float)(a[((tid+index+1)*size + j)] - (float)a[((tid+index+1)*size+index)] * a[((index*size) + j)]);
	}

}

int main(int argc,char **argv){
	float *dev_a;
	int i,p,q,n=0;
	struct timeval bef,aft;
 	long duration=0;

	n = atoi(argv[1]); // obtain the size of matrix

A = (float *) malloc(sizeof(float) * n * n);
L = (float *) malloc(sizeof(float) * n * n);
U = (float *) malloc(sizeof(float) * n * n);
input = (float *) malloc(sizeof(float*) * n * n);

	
	//allocate the memory on the GPU
	hipMalloc ((void**)&dev_a,n * n * sizeof(float));
	
	arrayInit(n);	// initialize the array

	//copy the arrays 'a' and 'b' to the GPU
	hipMemcpy( dev_a, A, n * n * sizeof(float), hipMemcpyHostToDevice);
	gettimeofday(&bef,NULL);
	for(i = 0;i<n;i++) {
	scale<<<1, 1>>> (dev_a, n, i);
	reduce<<<n-i-1,1>>>(dev_a, n, i);
	}
	gettimeofday(&aft,NULL);
	duration = aft.tv_sec - bef.tv_sec;
 	printf("%ld --- %d  \n",duration,n);

	//copy the array 'c' back from the GPU to the CPU
	hipMemcpy( A, dev_a, n * n * sizeof(float),hipMemcpyDeviceToHost );
								
	//update the array and display the results

	printf("\n");	
	updateLU(n);

	//free the memory allocated on the GPU
	hipFree( dev_a );
	
	return 0;
}
void updateLU(int n) {
	int i=0,j=0;
	for(i=0;i<n;i++) {
		for(j=i+1;j<n;j++) {
			U[i*n + j] = A[i*n + j];
		}
	}
	for(i=0;i<n;i++) {
		for(j=0;j<i+1;j++) {
			L[i*n + j] = A[i*n + j];
		}
	}

	verifyLU(n);
}
void arrayInit(int n) {
	int i=0,j=0;

	/* Initialize the Random Number Generator*/

	for(i=0;i<n;i++) {
		for(j=0;j<n;j++) {
			A[i*n + j] = (rand() % 5) + 1.0;
			input[i*n + j] = A[i*n + j];
			L[i*n + j] = 0.0f;
			if(i == j) {
				U[i*n + j] = 1.0f;
			}
			else {
				U[i*n + j] = 0.0f;
			}

	}
	}
}
/*
 * Performs the Multiplication of Lower and Upper Matricies and verify
 * the result of the reconstructed Matrix.
 */
void verifyLU(int n) {
int i=0,j=0,k=0;
float sum=0,error=0;
for(i=0;i<n;i++) {

	for(j=0;j<n;j++) {
		for(k=0;k<n;k++) {
			sum += L[i*n + k]*U[k*n + j];
		}
		A[i*n + j] = sum;
		error += input[i*n + j] - A[i*n + j];
		sum=0;
	}
}
//printf(" The error is %lf \n",error);
/* PRINT OUT VERIFIED MATRIX */
 printf("\n REST MATX \n");
/*
for(i=0;i<n;i++) {
	for(j=0;j<n;j++) {
		printf("%lf  ",A[i*n + j]);
	}
	printf("\n");
}

for(i=0;i<n;i++) {
	for(j=0;j<n;j++) {
		printf("%lf  ",input[i*n + j]);
	}
	printf("\n");
}
*/
if(error != error || error < 1 || error > -1) {
printf("Success \n”);
}
}
void freemem(int n) {
  int i=0;
for (i = 0; i < n; i++) {
  float * pt = A;
float * ptl = L;
float * ptu = U;  
 free(pt);
 free(ptl);
 free(ptu); 
	}

}
